#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 
#include <stdio.h>
 
const int N = 5; 
const int blocksize = 5; 
 
__global__ 
void hello(char *b) 
{
  switch(threadIdx.x){
    case 0: b[threadIdx.x] = 'W'; break;
    case 1: b[threadIdx.x] = 'o'; break;
    case 2: b[threadIdx.x] = 'r'; break;
    case 3: b[threadIdx.x] = 'l'; break;
    case 4: b[threadIdx.x] = 'd'; break;
    default: break;
  }
}
 
int main()
{
	char a[N] = "Hello";
	char b[N];
 
	char *bd;
	const int csize = N*sizeof(char);
 
	printf("%s", a);
 
	hipMalloc( (void**)&bd, csize ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(bd);
	hipMemcpy( b, bd, csize, hipMemcpyDeviceToHost ); 
	hipFree( bd );
	
	printf("%s %s\n", a, b);
	return EXIT_SUCCESS;
}
